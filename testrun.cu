
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
	printf("hello ");
}


#define N 512
int main(void) {
	
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N*sizeof(int);
	
	// host copies of a, b, c
	// device copies of a, b, c
	// Allocate space for device copies of a, c, b
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	// Setup input values
	a = (int *) malloc(size);
	b = (int *) malloc(size);

	for(int i=0; i<N; i++){
		a[i]=rand()%10;
		b[i]=rand()%10;
	}
	c = (int *) malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// Launch add() kernel on GPU
	add<<<N,1>>>(d_a, d_b, d_c);
	
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	for(int i=0; i<10; i++){
		printf("Executed: %d + %d = %d\n", a[i], b[i], c[i]);
	}

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	

	return 0;
}